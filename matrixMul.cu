// This code is written while learning CUDA from YT.
// Author : Vidnyani Umathe
// Date   : 01/12/2023
// Matrix Multiplication


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <algorithm>
#include <functional>

// STL
using std::cout;
using std::generate;
using std::vector;

__global__ void matrixMul(const int *a, const int *b, int *c, int N)
{

    // compute row and column index for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // iterate over row and column and put the result in c
    c[row * N + col] = 0;
    for (int i = 0; i < N; i++)
    {
        c[row * N + col] += a[row * N + i] * b[i * N + col];
    }
}

int main()
{

    int N = 1 << 10; // 1024 x  1024 matrix size

    size_t bytes = N * N * sizeof(int);

    // host vector
    vector<int> h_a(N * N);
    vector<int> h_b(N * N);
    vector<int> h_c(N * N);

    // initialize matrices
    generate(h_a.begin(), h_a.end(), []()
             { return rand() % 100; });
    generate(h_b.begin(), h_b.end(), []()
             { return rand() % 100; });

    // allocate device memory
    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // copy data from host to device
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    int THREADS = 32;

    int BLOCKS = N / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    // launch kernel
    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

    cout << "COMPLETED SUCCESSFULLY\n";

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}