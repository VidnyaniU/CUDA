// This code is written while learning CUDA from YT.
// Author : Vidnyani Umathe
// Date   : 02/12/2023
// Matrix Multiplication by coalescing the memory so as to minimize the time in accessing the memory
// Here to coalesce we just did a transpose of the matrix


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <vector>

using namespace std;
using std::cout;
using std::vector;

__global__ void matrixMul(const int *a, const int *b, int *c, int N)
{
    // compute row and column index for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // iterate over row and column and put the result in c
    c[row * N + col] = 0;
    if ((row < N) && (col < N))
    {
        for (int i = 0; i < N; i++)
        {
            c[row * N + col] += a[i * N + row] * b[i * N + col]; // takign transpose
        }
    }
}
// void transpose(int *a, int *a_t, int n)
// {
//     for (int i = 0; i < n; i++)
//     {
//         for (int j = 0; j < n; j++)
//         {
//             a_t[j * n + i] = a[i * n + j];
//         }
//     }
// }
int main()
{

    int N = 1024; // 1024 x  1024 matrix size
    size_t bytes = N * N * sizeof(int);

    // host vector
    vector<int> h_a;
    vector<int> h_b;
    vector<int> h_c(N * N);

    // initialize matrices
    for (int i = 0; i < N * N; i++)
    {
        h_a.push_back(1);
        h_b.push_back(1);
    }

    // allocate device memory
    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // copy data from host to device
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    int THREADS = 32;

    int BLOCKS = N / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    // launch kernel
    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

    // Copy back to the host
    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    cout << "COMPLETED SUCCESSFULLY\n";

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}